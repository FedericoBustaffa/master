
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello()
{
	printf("Hello CUDA\n");
	return;
}

int main(int argc, const char** argv)
{
	hello<<<1, 1>>>();
	hipDeviceSynchronize();
	return 0;
}